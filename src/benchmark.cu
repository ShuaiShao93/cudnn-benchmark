//
// Created by slimakanzer on 29.03.19.
//
#include <assert.h>
#include <chrono>
#include <stdexcept>
#include <iostream>
#include "benchmark.hpp"
#include "parser.hpp"

template<typename T, typename O>
void Benchmark<T, O>::create_cudnn() {
    CHECK_CUDNN_ERROR(hipdnnCreate(&cudnn));
}

template<typename T, typename O>
void Benchmark<T, O>::create_curand_generator() {
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);
}

template<typename T, typename O>
Benchmark<T, O>::Benchmark() {
    create_cudnn();
    create_curand_generator();
}

template<typename T, typename O>
size_t Benchmark<T, O>::fwd_workspace_size(hipdnnConvolutionFwdAlgo_t algo) {
    assert(cudnn);
    assert(inputTensorDescriptor);
    assert(filterDescriptor);
    assert(outputTensorDescriptor);

    size_t workspace_size = 0;
    CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                              inputTensorDescriptor->descriptor(),
                                                              filterDescriptor->descriptor(),
                                                              convolutionDescriptor_,
                                                              outputTensorDescriptor->descriptor(),
                                                              algo,
                                                              &workspace_size));
    return workspace_size;
}

template<typename T, typename O>
benchmarkResult Benchmark<T, O>::forward(hipdnnConvolutionFwdAlgo_t algo, uint32_t num_repeats) {
    assert(inputTensor);
    assert(outputTensor);
    assert(kernelTensor);

    size_t workspace_size;
    try {
        workspace_size = fwd_workspace_size(algo);
    } catch (std::exception &exception) {
        std::cerr << "WORKSPACE SIZE: " << get_fwd_algo_name(algo) << " " << exception.what();
        return {0, 0, BENCHMARK_NOT_SUPPORTED};
    }

    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_size);

    double fwd_time = 0;
    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < num_repeats; ++i) {
        hipdnnStatus_t
                fwd_status = hipdnnConvolutionForward(cudnn,
                                                     &alpha,
                                                     inputTensorDescriptor->descriptor(),
                                                     inputTensor->begin(),
                                                     filterDescriptor->descriptor(),
                                                     kernelTensor->begin(),
                                                     convolutionDescriptor_,
                                                     algo,
                                                     d_workspace,
                                                     workspace_size,
                                                     &beta,
                                                     outputTensorDescriptor->descriptor(),
                                                     outputTensor->begin());

        if (fwd_status != HIPDNN_STATUS_SUCCESS) {
            std::cerr << "CONVOLUTION: CUDNN failure: " << hipdnnGetErrorString(fwd_status) << "algo: " << get_fwd_algo_name(algo)
                      << " file: " << __FILE__ << " line: " << __LINE__ << std::endl;
            return {0, workspace_size, BENCHMARK_ERROR};
        }
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    fwd_time = std::chrono::duration<double, std::micro>(end - start).count() / num_repeats;
    hipFree(d_workspace);

    return {fwd_time, workspace_size, BENCHMARK_SUCCESS};
}

template<typename T, typename O>
void Benchmark<T, O>::forward_algorythms(uint32_t num_repeats) {
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_GEMM = forward(HIPDNN_CONVOLUTION_FWD_ALGO_GEMM, num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM = forward(HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                                                      num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM = forward(
            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT = forward(HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT, num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_FFT = forward(HIPDNN_CONVOLUTION_FWD_ALGO_FFT, num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING = forward(HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING, num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD = forward(HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD, num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED = forward(HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                                                          num_repeats);
}

template<typename T, typename O>
void Benchmark<T, O>::calculate_workspace_benchmark(uint32_t num_repeats) {
    assert(inputTensorDescriptor);
    assert(outputTensorDescriptor);
    assert(filterDescriptor);

    auto formatInputTensor = inputTensorDescriptor->format();
    auto formatOutputTensor = outputTensorDescriptor->format();
    auto formatFilter = filterDescriptor->format();

    inputTensor = new Tensor<T>(
            {formatInputTensor.N, formatInputTensor.H, formatInputTensor.W, formatInputTensor.C});
    outputTensor = new Tensor<O>(
            {formatOutputTensor.N, formatOutputTensor.H, formatOutputTensor.W, formatOutputTensor.C});
    kernelTensor = new Tensor<T>({formatFilter.N, formatFilter.H, formatFilter.W, formatFilter.C});

    inputTensor->rand(curand_gen);
    kernelTensor->rand(curand_gen);

    forward_algorythms(num_repeats);

    delete inputTensor;
    delete outputTensor;
    delete kernelTensor;
}

template<typename T, typename O>
void Benchmark<T, O>::benchmark(benchmarkRow &benchmarkInput, uint32_t num_repeats) {
    this->benchmark_row = &benchmarkInput;

    hipdnnDataType_t dataType;
    if (std::is_same<T, DATA_FLOAT>::value) {
        dataType = HIPDNN_DATA_FLOAT;
    } else if (std::is_same<T, DATA_DOUBLE>::value) {
        dataType = HIPDNN_DATA_DOUBLE;
    } else if (std::is_same<T, DATA_HALF_FLOAT>::value) {
        dataType = HIPDNN_DATA_HALF;
    } else if (std::is_same<T, DATA_INT32>::value) {
        dataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<T, DATA_INT8>::value) {
        dataType = HIPDNN_DATA_INT8;
    } else if (std::is_same<T, DATA_UINT8>::value) {
        dataType = CUDNN_DATA_UINT8;
    } else if (std::is_same<T, DATA_INT8x4>::value) {
        dataType = HIPDNN_DATA_INT8x4;
    } else if (std::is_same<T, DATA_INT8x32>::value) {
        dataType = CUDNN_DATA_INT8x32;
    } else if (std::is_same<T, DATA_UINT8x4>::value) {
        dataType = CUDNN_DATA_UINT8x4;
    } else {
        throw new std::runtime_error("Cannot find supported format");
    }

    hipdnnDataType_t computeDataType;
    if (std::is_same<O, DATA_FLOAT>::value) {
        computeDataType = HIPDNN_DATA_FLOAT;
    } else if (std::is_same<O, DATA_DOUBLE>::value) {
        computeDataType = HIPDNN_DATA_DOUBLE;
    } else if (std::is_same<O, DATA_HALF_FLOAT>::value) {
        computeDataType = HIPDNN_DATA_HALF;
    } else if (std::is_same<O, DATA_INT32>::value) {
        computeDataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<O, DATA_INT8>::value) {
        computeDataType = HIPDNN_DATA_INT8;
    } else if (std::is_same<O, DATA_UINT8>::value) {
        computeDataType = CUDNN_DATA_UINT8;
    } else if (std::is_same<O, DATA_INT8x4>::value) {
        computeDataType = HIPDNN_DATA_INT8x4;
    } else if (std::is_same<O, DATA_INT8x32>::value) {
        computeDataType = CUDNN_DATA_INT8x32;
    } else if (std::is_same<O, DATA_UINT8x4>::value) {
        computeDataType = CUDNN_DATA_UINT8x4;
    } else {
        throw new std::runtime_error("Cannot find supported format");
    }

    Format formatInputTensor = {
            benchmarkInput.n,
            benchmarkInput.c,
            benchmarkInput.h,
            benchmarkInput.w,
            benchmarkInput.inputTensorFormat
    };

    Format formatOutputTensor = {
            benchmarkInput.n,
            benchmarkInput.k,
            benchmarkInput.out_h,
            benchmarkInput.out_w,
            benchmarkInput.outputTensorFormat
    };

    Format formatFilter = {
            benchmarkInput.k,
            benchmarkInput.c,
            benchmarkInput.r,
            benchmarkInput.s,
            benchmarkInput.filterFormat
    };

    inputTensorDescriptor = new TensorDescriptor(formatInputTensor, dataType);
    outputTensorDescriptor = new TensorDescriptor(formatOutputTensor, dataType);
    filterDescriptor = new FilterDescriptor(formatFilter, dataType);


    CHECK_CUDNN_ERROR(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor_));

    CHECK_CUDNN_ERROR(hipdnnSetConvolution2dDescriptor(convolutionDescriptor_,
                                                      benchmarkInput.pad_h,
                                                      benchmarkInput.pad_w,
                                                      benchmarkInput.stride_h,
                                                      benchmarkInput.stride_w,
                                                      1,
                                                      1,
                                                      HIPDNN_CROSS_CORRELATION,
                                                      computeDataType));
    int n, c, h, w;

    CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(
            convolutionDescriptor_,
            inputTensorDescriptor->descriptor(),
            filterDescriptor->descriptor(),
            &n,
            &c,
            &h,
            &w));

    std::cerr << "OUT VALUES: " << h <<" " << w << " " << c << " " << n << std::endl;

    hipdnnSetConvolutionMathType(convolutionDescriptor_, HIPDNN_TENSOR_OP_MATH);

    calculate_workspace_benchmark(num_repeats);

    delete inputTensorDescriptor;
    delete outputTensorDescriptor;
    delete filterDescriptor;

    CHECK_CUDNN_ERROR(hipdnnDestroyConvolutionDescriptor(convolutionDescriptor_));
}

template<typename T, typename O>
void
Benchmark<T, O>::run(std::string file_name, std::string output_file_name, bool all_formats,
                  uint32_t num_repeats,
                  hipdnnTensorFormat_t input_format, hipdnnTensorFormat_t output_format,
                  hipdnnTensorFormat_t kernel_format) {

    auto benchmark_rows = parser::readInputDataFile(file_name);

    Benchmark<T, O> benchmark;
    parser::Parser<T, O> parser(&benchmark, output_file_name);
    for (auto row : benchmark_rows) {
        if (!all_formats) {
            row.inputTensorFormat = input_format;
            row.outputTensorFormat = output_format;
            row.filterFormat = kernel_format;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << std::endl;
            }
        } else {
            row.inputTensorFormat = HIPDNN_TENSOR_NCHW;
            row.outputTensorFormat = HIPDNN_TENSOR_NCHW;
            row.filterFormat = HIPDNN_TENSOR_NCHW;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << std::endl;
            }

            row.inputTensorFormat = HIPDNN_TENSOR_NHWC;
            row.outputTensorFormat = HIPDNN_TENSOR_NHWC;
            row.filterFormat = HIPDNN_TENSOR_NHWC;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << std::endl;
            }

            row.inputTensorFormat = HIPDNN_TENSOR_NCHW_VECT_C;
            row.outputTensorFormat = HIPDNN_TENSOR_NCHW_VECT_C;
            row.filterFormat = HIPDNN_TENSOR_NCHW_VECT_C;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << "THIS FORMAT NOT SUPPORT CURRENT DATA TYPE" << std::endl;
            }
        }
    }
    parser.closeOutFile();
}

int main(int argc, char **argv) {
    if (argc < 5) {
        std::cerr << "ERROR ARGS PROGRAM: \n"
                     "file_name - name of input file with convolution cases\n"
                     "file_name_output - name of output file with benchmark result\n"
                     "data_type - type of data values (like fp16 and etc)\n"
                     "all_format - use all cudnn data format (true/false)\n"
                     "num_repeats - number of repeats per one algorithm\n"
                     "input_tensor_data_format - format of input tensor\n"
                     "output_tensor_data_format - format of output tensor\n"
                     "kernel_tensor_data_format - format of kernel tensor\n" << std::endl;
        return 1;

    }

    std::string file_name = argv[1];
    std::string output_file_name = argv[2];
    std::string data_type_name = argv[3];
    bool all_formats = static_cast<bool>(std::stoi(argv[4]));
    uint32_t num_repeats = static_cast<uint32_t>(std::stoi(argv[5]));

    if (!all_formats && (argc < 9)) {
        std::cerr << "input_tensor_data_format - format of input tensor\n"
                     "output_tensor_data_format - format of output tensor\n"
                     "kernel_tensor_data_format - format of kernel tensor\n" << std::endl;
        return 1;
    }

    hipdnnTensorFormat_t input_format;
    hipdnnTensorFormat_t output_format;
    hipdnnTensorFormat_t kernel_format;
    if (!all_formats) {
        input_format = get_data_format_by_name(argv[6]);
        output_format = get_data_format_by_name(argv[7]);
        kernel_format = get_data_format_by_name(argv[8]);
    }

    if (data_type_name.compare("fp16") == 0)
        Benchmark<DATA_HALF_FLOAT, DATA_HALF_FLOAT>::run(file_name, output_file_name, all_formats, num_repeats,
                                        input_format, output_format,
                                        kernel_format);
    else if (data_type_name.compare("fp32") == 0)
        Benchmark<DATA_FLOAT>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                   output_format,
                                   kernel_format);
    else if (data_type_name.compare("fp64") == 0)
        Benchmark<DATA_DOUBLE>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                    output_format,
                                    kernel_format);
    else if (data_type_name.compare("int8") == 0)
        Benchmark<DATA_INT8, DATA_INT32>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                  output_format,
                                  kernel_format);
    else if (data_type_name.compare("uint8") == 0)
        Benchmark<DATA_UINT8>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                   output_format,
                                   kernel_format);
    else if (data_type_name.compare("int32") == 0)
        Benchmark<DATA_INT32>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                   output_format,
                                   kernel_format);
    else if (data_type_name.compare("int8x4") == 0)
        Benchmark<DATA_INT8x4>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                    output_format,
                                    kernel_format);
    else if (data_type_name.compare("int8x32") == 0)
        Benchmark<DATA_INT8x32>::run(file_name, output_file_name, all_formats, num_repeats,
                                     input_format, output_format,
                                     kernel_format);
    else if (data_type_name.compare("uint8x4") == 0)
        Benchmark<DATA_UINT8x4>::run(file_name, output_file_name, all_formats, num_repeats,
                                     input_format, output_format,
                                     kernel_format);
    else std::cerr << "Data type not supported" << std::endl;

    return 0;
}