//
// Created by slimakanzer on 29.03.19.
//
#include <assert.h>
#include <chrono>
#include <stdexcept>
#include <iostream>
#include "benchmark.hpp"
#include "parser.hpp"

template<typename T>
void Benchmark<T>::create_cudnn() {
    CHECK_CUDNN_ERROR(hipdnnCreate(&cudnn));
}

template<typename T>
void Benchmark<T>::create_curand_generator() {
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);
}

template<typename T>
Benchmark<T>::Benchmark() {
    create_cudnn();
    create_curand_generator();
}

template<typename T>
size_t Benchmark<T>::fwd_workspace_size(hipdnnConvolutionFwdAlgo_t algo) {
    assert(cudnn);
    assert(inputTensorDescriptor);
    assert(filterDescriptor);
    assert(outputTensorDescriptor);

    size_t workspace_size = 0;
    CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                              inputTensorDescriptor->descriptor(),
                                                              filterDescriptor->descriptor(),
                                                              convolutionDescriptor_,
                                                              outputTensorDescriptor->descriptor(),
                                                              algo,
                                                              &workspace_size));
    return workspace_size;
}

template<typename T>
benchmarkResult Benchmark<T>::forward(hipdnnConvolutionFwdAlgo_t algo, uint32_t num_repeats) {
    assert(inputTensor);
    assert(outputTensor);
    assert(kernelTensor);

    size_t workspace_size;
    try {
        workspace_size = fwd_workspace_size(algo);
    } catch (std::exception &exception) {
        std::cerr << "WORKSPACE SIZE: " << get_fwd_algo_name(algo) << " " << exception.what();
        return {0, 0, BENCHMARK_NOT_SUPPORTED};
    }

    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_size);

    double fwd_time = 0;
    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < num_repeats; ++i) {
        hipdnnStatus_t
                fwd_status = hipdnnConvolutionForward(cudnn,
                                                     &alpha,
                                                     inputTensorDescriptor->descriptor(),
                                                     inputTensor->begin(),
                                                     filterDescriptor->descriptor(),
                                                     kernelTensor->begin(),
                                                     convolutionDescriptor_,
                                                     algo,
                                                     d_workspace,
                                                     workspace_size,
                                                     &beta,
                                                     outputTensorDescriptor->descriptor(),
                                                     outputTensor->begin());

        if (fwd_status != HIPDNN_STATUS_SUCCESS) {
            std::cerr << "CONVOLUTION: CUDNN failure: " << hipdnnGetErrorString(fwd_status) << "algo: " << get_fwd_algo_name(algo)
                      << " file: " << __FILE__ << " line: " << __LINE__ << std::endl;
            return {0, workspace_size, BENCHMARK_ERROR};
        }
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    fwd_time = std::chrono::duration<double, std::micro>(end - start).count() / num_repeats;
    hipFree(d_workspace);

    return {fwd_time, workspace_size, BENCHMARK_SUCCESS};
}

template<typename T>
void Benchmark<T>::forward_algorythms(uint32_t num_repeats) {
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_GEMM = forward(HIPDNN_CONVOLUTION_FWD_ALGO_GEMM, num_repeats);
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM = forward(HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
    //                                                                   num_repeats);
    benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM = forward(
            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, num_repeats);
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT = forward(HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT, num_repeats);
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_FFT = forward(HIPDNN_CONVOLUTION_FWD_ALGO_FFT, num_repeats);
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING = forward(HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING, num_repeats);
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD = forward(HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD, num_repeats);
    // benchmark_row->HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED = forward(HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
    //                                                                       num_repeats);
}

template<typename T>
void Benchmark<T>::calculate_workspace_benchmark(uint32_t num_repeats) {
    assert(inputTensorDescriptor);
    assert(outputTensorDescriptor);
    assert(filterDescriptor);

    auto formatInputTensor = inputTensorDescriptor->format();
    auto formatOutputTensor = outputTensorDescriptor->format();
    auto formatFilter = filterDescriptor->format();

    inputTensor = new Tensor<T>(
            {formatInputTensor.N, formatInputTensor.H, formatInputTensor.W, formatInputTensor.C});
    outputTensor = new Tensor<T>(
            {formatOutputTensor.N, formatOutputTensor.H, formatOutputTensor.W, formatOutputTensor.C});
    auto origKernelTensor = new Tensor<T>({formatFilter.N, formatFilter.H, formatFilter.W, formatFilter.C});
    kernelTensor = new Tensor<T>({formatFilter.N, formatFilter.H, formatFilter.W, formatFilter.C});

    inputTensor->rand(curand_gen);
    origKernelTensor->rand(curand_gen);
    if (filterDescriptor->dataType() == CUDNN_DATA_INT8x32) {
        CHECK_CUDNN_ERROR(cudnnReorderFilterAndBias(
            cudnn, filterDescriptor->descriptor(), CUDNN_DEFAULT_REORDER, origKernelTensor->begin(), kernelTensor->begin(), false, NULL, NULL));

        CHECK_CUDNN_ERROR(cudnnSetConvolutionReorderType(convolutionDescriptor_, CUDNN_NO_REORDER));
    } else {
        hipMemcpy(kernelTensor->begin(), origKernelTensor->begin(), sizeof(origKernelTensor->begin()[0]) * origKernelTensor->size(), hipMemcpyDeviceToDevice);
    }

    forward_algorythms(num_repeats);

    delete inputTensor;
    delete outputTensor;
    delete origKernelTensor;
    delete kernelTensor;
}

template<typename T>
void Benchmark<T>::benchmark(benchmarkRow &benchmarkInput, uint32_t num_repeats) {
    this->benchmark_row = &benchmarkInput;

    hipdnnDataType_t dataType;
    hipdnnDataType_t computeDataType;
    if (std::is_same<T, DATA_FLOAT>::value) {
        dataType = HIPDNN_DATA_FLOAT;
        computeDataType = HIPDNN_DATA_FLOAT;
    } else if (std::is_same<T, DATA_DOUBLE>::value) {
        dataType = HIPDNN_DATA_DOUBLE;
        computeDataType = HIPDNN_DATA_DOUBLE;
    } else if (std::is_same<T, DATA_HALF_FLOAT>::value) {
        dataType = HIPDNN_DATA_HALF;
        computeDataType = HIPDNN_DATA_HALF;
    } else if (std::is_same<T, DATA_INT32>::value) {
        dataType = HIPDNN_DATA_INT32;
        computeDataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<T, DATA_INT8>::value) {
        dataType = HIPDNN_DATA_INT8;
        computeDataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<T, DATA_UINT8>::value) {
        dataType = CUDNN_DATA_UINT8;
        computeDataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<T, DATA_INT8x4>::value) {
        dataType = HIPDNN_DATA_INT8x4;
        computeDataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<T, DATA_INT8x32>::value) {
        dataType = CUDNN_DATA_INT8x32;
        computeDataType = HIPDNN_DATA_INT32;
    } else if (std::is_same<T, DATA_UINT8x4>::value) {
        dataType = CUDNN_DATA_UINT8x4;
        computeDataType = HIPDNN_DATA_INT32;
    } else {
        throw new std::runtime_error("Cannot find supported format");
    }

    Format formatInputTensor = {
            benchmarkInput.n,
            benchmarkInput.c,
            benchmarkInput.h,
            benchmarkInput.w,
            benchmarkInput.inputTensorFormat
    };

    Format formatOutputTensor = {
            benchmarkInput.n,
            benchmarkInput.k,
            benchmarkInput.out_h,
            benchmarkInput.out_w,
            benchmarkInput.outputTensorFormat
    };

    Format formatFilter = {
            benchmarkInput.k,
            benchmarkInput.c,
            benchmarkInput.r,
            benchmarkInput.s,
            benchmarkInput.filterFormat
    };

    inputTensorDescriptor = new TensorDescriptor(formatInputTensor, dataType);
    outputTensorDescriptor = new TensorDescriptor(formatOutputTensor, dataType);
    filterDescriptor = new FilterDescriptor(formatFilter, dataType);


    CHECK_CUDNN_ERROR(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor_));
    CHECK_CUDNN_ERROR(hipdnnSetConvolution2dDescriptor(convolutionDescriptor_,
                                                      benchmarkInput.pad_h,
                                                      benchmarkInput.pad_w,
                                                      benchmarkInput.stride_h,
                                                      benchmarkInput.stride_w,
                                                      1,
                                                      1,
                                                      HIPDNN_CROSS_CORRELATION,
                                                      computeDataType));
    int n, c, h, w;

    CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(
            convolutionDescriptor_,
            inputTensorDescriptor->descriptor(),
            filterDescriptor->descriptor(),
            &n,
            &c,
            &h,
            &w));

    if (n != formatOutputTensor.N || c != formatOutputTensor.C ||
        h != formatOutputTensor.H || w != formatOutputTensor.W) {
            throw std::runtime_error("Wrong output shape.");
    }

    std::cerr << "OUT VALUES: " << h <<" " << w << " " << c << " " << n << std::endl;

    hipdnnSetConvolutionMathType(convolutionDescriptor_, HIPDNN_TENSOR_OP_MATH);

    calculate_workspace_benchmark(num_repeats);

    delete inputTensorDescriptor;
    delete outputTensorDescriptor;
    delete filterDescriptor;

    CHECK_CUDNN_ERROR(hipdnnDestroyConvolutionDescriptor(convolutionDescriptor_));
}

template<typename T>
void
Benchmark<T>::run(std::string file_name, std::string output_file_name, bool all_formats,
                  uint32_t num_repeats,
                  hipdnnTensorFormat_t input_format, hipdnnTensorFormat_t output_format,
                  hipdnnTensorFormat_t kernel_format) {

    auto benchmark_rows = parser::readInputDataFile(file_name);

    Benchmark<T> benchmark;
    parser::Parser<T> parser(&benchmark, output_file_name);
    for (auto row : benchmark_rows) {
        if (!all_formats) {
            row.inputTensorFormat = input_format;
            row.outputTensorFormat = output_format;
            row.filterFormat = kernel_format;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << std::endl;
            }
        } else {
            row.inputTensorFormat = HIPDNN_TENSOR_NCHW;
            row.outputTensorFormat = HIPDNN_TENSOR_NCHW;
            row.filterFormat = HIPDNN_TENSOR_NCHW;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << std::endl;
            }

            row.inputTensorFormat = HIPDNN_TENSOR_NHWC;
            row.outputTensorFormat = HIPDNN_TENSOR_NHWC;
            row.filterFormat = HIPDNN_TENSOR_NHWC;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << std::endl;
            }

            row.inputTensorFormat = HIPDNN_TENSOR_NCHW_VECT_C;
            row.outputTensorFormat = HIPDNN_TENSOR_NCHW_VECT_C;
            row.filterFormat = HIPDNN_TENSOR_NCHW_VECT_C;

            try {
                benchmark.benchmark(row, num_repeats);
                parser.writeBenchmarkResult();
            } catch (std::exception &e) {
                std::cerr << "Exception: " << e.what() << "THIS FORMAT NOT SUPPORT CURRENT DATA TYPE" << std::endl;
            }
        }
    }
    parser.closeOutFile();
}

int main(int argc, char **argv) {
    if (argc < 5) {
        std::cerr << "ERROR ARGS PROGRAM: \n"
                     "file_name - name of input file with convolution cases\n"
                     "file_name_output - name of output file with benchmark result\n"
                     "data_type - type of data values (like fp16 and etc)\n"
                     "all_format - use all cudnn data format (true/false)\n"
                     "num_repeats - number of repeats per one algorithm\n"
                     "input_tensor_data_format - format of input tensor\n"
                     "output_tensor_data_format - format of output tensor\n"
                     "kernel_tensor_data_format - format of kernel tensor\n" << std::endl;
        return 1;

    }

    std::string file_name = argv[1];
    std::string output_file_name = argv[2];
    std::string data_type_name = argv[3];
    bool all_formats = static_cast<bool>(std::stoi(argv[4]));
    uint32_t num_repeats = static_cast<uint32_t>(std::stoi(argv[5]));

    if (!all_formats && (argc < 9)) {
        std::cerr << "input_tensor_data_format - format of input tensor\n"
                     "output_tensor_data_format - format of output tensor\n"
                     "kernel_tensor_data_format - format of kernel tensor\n" << std::endl;
        return 1;
    }

    hipdnnTensorFormat_t input_format;
    hipdnnTensorFormat_t output_format;
    hipdnnTensorFormat_t kernel_format;
    if (!all_formats) {
        input_format = get_data_format_by_name(argv[6]);
        output_format = get_data_format_by_name(argv[7]);
        kernel_format = get_data_format_by_name(argv[8]);
    }

    if (data_type_name.compare("fp16") == 0)
        Benchmark<DATA_HALF_FLOAT>::run(file_name, output_file_name, all_formats, num_repeats,
                                        input_format, output_format,
                                        kernel_format);
    else if (data_type_name.compare("fp32") == 0)
        Benchmark<DATA_FLOAT>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                   output_format,
                                   kernel_format);
    else if (data_type_name.compare("fp64") == 0)
        Benchmark<DATA_DOUBLE>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                    output_format,
                                    kernel_format);
    else if (data_type_name.compare("int8") == 0)
        Benchmark<DATA_INT8>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                  output_format,
                                  kernel_format);
    else if (data_type_name.compare("uint8") == 0)
        Benchmark<DATA_UINT8>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                   output_format,
                                   kernel_format);
    else if (data_type_name.compare("int32") == 0)
        Benchmark<DATA_INT32>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                   output_format,
                                   kernel_format);
    else if (data_type_name.compare("int8x4") == 0)
        Benchmark<DATA_INT8x4>::run(file_name, output_file_name, all_formats, num_repeats, input_format,
                                    output_format,
                                    kernel_format);
    else if (data_type_name.compare("int8x32") == 0)
        Benchmark<DATA_INT8x32>::run(file_name, output_file_name, all_formats, num_repeats,
                                     input_format, output_format,
                                     kernel_format);
    else if (data_type_name.compare("uint8x4") == 0)
        Benchmark<DATA_UINT8x4>::run(file_name, output_file_name, all_formats, num_repeats,
                                     input_format, output_format,
                                     kernel_format);
    else std::cerr << "Data type not supported" << std::endl;

    return 0;
}